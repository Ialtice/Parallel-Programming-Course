
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// __global__ keyword specifies a device kernel function
__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
    printf("hello form GPU B.x=%d, Thread.x=%d\n", blockIdx.x, threadIdx.x);
}

int main() {
    printf("Hello World from CPU!\n");

    // Call a device function from the host: a kernel launch Which will print from the device
    cuda_hello<<<6,1>>>(); 
	//cuda_hello<<<1,6>>>();  comment and uncomment lines 13-14 for lab step 7 and 8
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    return 0; 
}

